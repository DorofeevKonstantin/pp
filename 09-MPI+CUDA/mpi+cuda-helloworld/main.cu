#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <math.h>

#include "mpi.h"

#include "hip/hip_runtime.h"
#include ""

void Error(hipError_t cudaStatus)
{
	if (cudaStatus != hipSuccess)
	{
		printf("Some Error : %s\n", hipGetErrorString(cudaStatus));
	}
}
__global__ void kernel()
{
	printf("kernel <<<(%d,%d,%d),(%d,%d,%d)>>>\n", gridDim.x, gridDim.y, gridDim.z, blockDim.x, blockDim.y, blockDim.z);
}
void DeviceOutput(hipDeviceProp_t& prop)
{
	printf("CUDA Device info:\n");
	printf("Name: %s\n", prop.name);
	printf("Architecture: %d.%d\n", prop.major, prop.minor);
	printf("Clockrate: %d\n", prop.clockRate);
	printf("Globalmem: %ld\n", prop.totalGlobalMem);
	printf("SM: %d\n", prop.multiProcessorCount);
	printf("WarpSize: %d\n", prop.warpSize);
	printf("ThreadsPerBlock: %d\n", prop.maxThreadsPerBlock);
	printf("Grid Size: (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
}

int main(void)
{
	MPI_Init(0, 0);
	hipDeviceProp_t prop;
	int size, rank, length = 0, count;
	char name[100];
	MPI_Comm_rank(MPI_COMM_WORLD, &rank);
	MPI_Comm_size(MPI_COMM_WORLD, &size);
	MPI_Get_processor_name(name, &length);
	printf("process %d of %d is running on %s\n", rank, size, name);
	hipError_t cudaStatus;
	cudaStatus = hipGetDeviceCount(&count);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipGetDeviceCount failed!\n");
		return -1;
	}
	printf("%d CUDA devices was found\n", count);
	for (int i = 0; i < count; i++)
	{
		cudaStatus = hipGetDeviceProperties(&prop, i);
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipGetDeviceProperties failed!\n");
			return -1;
		}
		else
			DeviceOutput(prop);
	}
	kernel << <2, 2 >> > ();
	hipDeviceSynchronize();
	MPI_Finalize();
	return 0;
}