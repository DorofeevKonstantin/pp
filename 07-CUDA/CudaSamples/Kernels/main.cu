#include <stdio.h>
#include <stdlib.h>
#include <memory.h>

#include "hip/hip_runtime.h"
#include ""

#include "vectorSumm.cuh"

#define SIZE 10

void vectorSummExample()
{
	int a[SIZE], b[SIZE], c[SIZE];
	int* dev_a, * dev_b, * dev_c;
	hipMalloc((void**)&dev_a, SIZE * sizeof(int));
	hipMalloc((void**)&dev_b, SIZE * sizeof(int));
	hipMalloc((void**)&dev_c, SIZE * sizeof(int));
	for (int i = 0; i < SIZE; i++)
	{
		a[i] = i;
		b[i] = i * i;
	}
	hipMemcpy(dev_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, SIZE * sizeof(int), hipMemcpyHostToDevice);

	vectorSummThreadsKernel << <1, SIZE >> > (dev_a, dev_b, dev_c, SIZE);
	hipDeviceSynchronize();
	hipMemcpy(c, dev_c, SIZE * sizeof(int), hipMemcpyDeviceToHost);
	printVectorSumm(a, b, c, SIZE);

	vectorSummLongKernel << <2, 2 >> > (dev_a, dev_b, dev_c, SIZE);
	hipMemcpy(c, dev_c, SIZE * sizeof(int), hipMemcpyDeviceToHost);
	printVectorSumm(a, b, c, SIZE);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
}

int main()
{
	vectorSummExample();
	return 0;
}