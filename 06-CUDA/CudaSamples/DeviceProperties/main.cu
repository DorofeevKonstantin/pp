#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""

void printDeviceProperties(hipDeviceProp_t* prop)
{
	printf("CUDA Device info:\n");
	printf("Name: %s\n", prop->name);
	printf("Architecture: %d.%d\n", prop->major, prop->minor);
	printf("Clockrate: %d Mhz\n", prop->clockRate / 1000);
	printf("Globalmem: %zd Mbytes\n", prop->totalGlobalMem / (1024 * 1024));
	printf("SharedmemPerBlock: %zd Kbytes\n", prop->sharedMemPerBlock / 1024);
	printf("multiProcessors: %d\n", prop->multiProcessorCount);
	printf("WarpSize: %d\n", prop->warpSize);
	printf("ThreadsPerBlock: %d\n", prop->maxThreadsPerBlock);
	printf("Maximum Grid Size: (%d, %d, %d)\n", prop->maxGridSize[0], prop->maxGridSize[1], prop->maxGridSize[2]);
	printf("Maximum Block Size: (%d, %d, %d)\n", prop->maxThreadsDim[0], prop->maxThreadsDim[1], prop->maxThreadsDim[2]);
}

int main()
{
	hipDeviceProp_t deviceProperties;
	int devicesCount;
	hipError_t cudaStatus;
	cudaStatus = hipGetDeviceCount(&devicesCount);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipGetDeviceCount failed!\n");
		return -1;
	}
	printf("%d devices was found\n", devicesCount);
	for (int i = 0; i < devicesCount; i++)
	{
		cudaStatus = hipGetDeviceProperties(&deviceProperties, i);
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipGetDeviceProperties failed!\n");
			return -1;
		}
		else
			printDeviceProperties(&deviceProperties);
	}
	return 0;
}