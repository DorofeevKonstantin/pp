#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"
#include ""

#include "vectorSumm.cuh"

__global__ void simpleKernel(void)
{
	printf("i am %d %d %d  block\n", blockIdx.x, blockIdx.y, blockIdx.z);
	if (blockIdx.x == 0)
		printf(" -> threads count in block : %d\n", gridDim.x);
}
void simpleExample()
{
	simpleKernel << <5, 1 >> > ();
	hipDeviceSynchronize();
	printf("simpleKernel end\n");
}

__global__ void addIntegersKernel(int a, int b, int* c)
{
	*c = a + b;
}
void addIntegersExample()
{
	int c;
	int* dev_c;
	hipMalloc((void**)&dev_c, sizeof(int));
	addIntegersKernel << <2, 2 >> > (2, 7, dev_c);
	hipDeviceSynchronize();
	hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
	printf("addIntegersKernel(2,7) == %d\n", c);
	hipFree(dev_c);
}

void vectorSummExample()
{
	const int size = 10;
	int* a, * b, * c;
	a = (int*)malloc(size * sizeof(int));
	b = (int*)malloc(size * sizeof(int));
	c = (int*)malloc(size * sizeof(int));
	int* dev_a, * dev_b, * dev_c;
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	for (int i = 0; i < size; i++)
	{
		a[i] = -i;
		b[i] = i * i;
	}
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	vectorSummBlocksKernel << <size, 1 >> > (dev_c, dev_a, dev_b, size);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	printVectorSumm(a, b, c, size);
Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);
	free(a);
	free(b);
	free(c);
}

int main()
{
	simpleExample();
	addIntegersExample();
	vectorSummExample();
	return 0;
}