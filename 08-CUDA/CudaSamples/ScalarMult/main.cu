#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include ""

const int N = 1000;
const int threadsPerBlock = 256;
const int blocksPerGrid = 2;

void Error(hipError_t cudaStatus)
{
	if (cudaStatus != hipSuccess)
		printf("CUDA error : %s\n", hipGetErrorString(cudaStatus));
}
__global__ void scalarMultKernel(float* a, float* b, float* c)
{
	__shared__ float cache[threadsPerBlock];
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int cacheIndex = threadIdx.x;
	float temp = 0;
	while (tid < N)
	{
		temp += a[tid] * b[tid];
		tid += blockDim.x * gridDim.x;
	}
	cache[cacheIndex] = temp;
	__syncthreads();
	int i = blockDim.x / 2;
	while (i != 0)
	{
		if (cacheIndex < i)
			cache[cacheIndex] += cache[cacheIndex + i];
		__syncthreads();
		i /= 2;
	}
	if (cacheIndex == 0)
		c[blockIdx.x] = cache[0];
}

int main(void)
{
	float* a, * b, c, * partialC;
	float* devA, * devB, * devPartialC;
	a = (float*)malloc(N * sizeof(float));
	b = (float*)malloc(N * sizeof(float));
	partialC = (float*)malloc(blocksPerGrid * sizeof(float));
	Error(hipMalloc((void**)&devA, N * sizeof(float)));
	Error(hipMalloc((void**)&devB, N * sizeof(float)));
	Error(hipMalloc((void**)&devPartialC, blocksPerGrid * sizeof(float)));
	for (int i = 0; i < N; i++)
	{
		a[i] = 1;
		b[i] = i;
	}
	Error(hipMemcpy(devA, a, N * sizeof(float), hipMemcpyHostToDevice));
	Error(hipMemcpy(devB, b, N * sizeof(float), hipMemcpyHostToDevice));
	scalarMultKernel << <blocksPerGrid, threadsPerBlock >> > (devA, devB, devPartialC);
	Error(hipDeviceSynchronize());
	Error(hipMemcpy(partialC, devPartialC, blocksPerGrid * sizeof(float), hipMemcpyDeviceToHost));
	c = 0;
	for (int i = 0; i < blocksPerGrid; i++)
		c += partialC[i];
	printf("Value = %f\n", c);
	Error(hipFree(devA));
	Error(hipFree(devB));
	Error(hipFree(devPartialC));
	free(a);
	free(b);
	free(partialC);
}